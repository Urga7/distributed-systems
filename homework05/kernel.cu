
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C"
{
#endif

    __global__ void medianFilter(uint8_t* imgIn, uint8_t* imgOut, int width, int height) {
        const int windowSize = 3;
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (x >= width || y >= height) return;

        int halfWindow = windowSize / 2;
        uint8_t window[windowSize * windowSize];

        int count = 0;
        for (int wy = -halfWindow; wy <= halfWindow; ++wy) {
            for (int wx = -halfWindow; wx <= halfWindow; ++wx) {
                int nx = min(max(x + wx, 0), width - 1);
                int ny = min(max(y + wy, 0), height - 1);
                window[count++] = imgIn[ny * width + nx];
            }
        }

        for (int i = 0; i < count - 1; ++i) {
            for (int j = i + 1; j < count; ++j) {
                if (window[i] > window[j]) {
                    uint8_t temp = window[i];
                    window[i] = window[j];
                    window[j] = temp;
                }
            }
        }

        imgOut[y * width + x] = window[count / 2];
    }


#ifdef __cplusplus
}
#endif