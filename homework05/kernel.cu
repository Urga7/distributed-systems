
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif

__global__ void medianFilter(unsigned char* imgIn, unsigned char* imgOut, int width, int height) {
    const int windowSize = 3;
    const int halfWindow = windowSize / 2;

    // Thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Threads outside image boundary
    if (x >= width || y >= height) return;

    unsigned char window[windowSize * windowSize];
    int count = 0;

    for (int wy = -halfWindow; wy <= halfWindow; ++wy) {
        for (int wx = -halfWindow; wx <= halfWindow; ++wx) {
            int nx = min(max(x + wx, 0), width - 1);
            int ny = min(max(y + wy, 0), height - 1);
            window[count++] = imgIn[ny * width + nx];
        }
    }

    for (int i = 1; i < count; ++i) {
        unsigned char key = window[i];
        int j = i - 1;
        while (j >= 0 && window[j] > key) {
            window[j + 1] = window[j];
            j--;
        }
        window[j + 1] = key;
    }

    imgOut[y * width + x] = window[count / 2];
}

#ifdef __cplusplus
}
#endif
